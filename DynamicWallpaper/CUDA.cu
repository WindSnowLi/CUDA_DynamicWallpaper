#include "hip/hip_runtime.h"
#include "CUDA.cuh"
#include <stdio.h>
#include <Windows.h>
__global__ void CUDAWaveSpread(int* m_pWaveBuf1, int* m_pWaveBuf2, int m_iBmpWidth, int m_iBmpHeight) {
	int* lpWave1 = m_pWaveBuf1;
	int* lpWave2 = m_pWaveBuf2;
	int Bid = blockIdx.x;
	int Tid = threadIdx.x;
	for (int i = m_iBmpWidth+ Bid*64*m_iBmpWidth+Tid * m_iBmpWidth; i < m_iBmpWidth + Bid * 64*m_iBmpWidth + Tid * m_iBmpWidth + m_iBmpWidth; i++)
	{
		//������ɢ
		lpWave2[i] = ((lpWave1[i - 1] + lpWave1[i - m_iBmpWidth] +
			lpWave1[i + 1] + lpWave1[i + m_iBmpWidth]) >> 1) - lpWave2[i];

		//����˥��
		lpWave2[i] -= (lpWave2[i] >> 5);
	}
	
	
}

__global__ void CUDAWaveRender(int* m_pWaveBuf1, BYTE* m_pBmpRender, BYTE* m_pBmpSource, int m_iBytesPerWidth, int m_iBmpWidth, int m_iBmpHeight) {
	int iPtrSource = 0;
	int iPtrRender = 0;
	int iPosX = 0;
	int iPosY = 0;
	int y= blockIdx.x*64+threadIdx.x+1;
	int lineIndex = y * m_iBmpWidth;
	for (int x = 0; x < m_iBmpWidth; x++)
	{
		//���ݲ�������λͼ����ƫ��ֵ����Ⱦ�㣨x��y)��ӦԭʼͼƬ��iPosX��iPosY��
		iPosX = x + (m_pWaveBuf1[lineIndex - 1] - m_pWaveBuf1[lineIndex + 1]);
		iPosY = y + (m_pWaveBuf1[lineIndex - m_iBmpWidth] - m_pWaveBuf1[lineIndex + m_iBmpWidth]);
		//����һ�ּ���ƫ�Ƶķ���
		//int waveData = (1024 - m_pWaveBuf1[lineIndex]);
		//iPosX = (x - m_iBmpWidth/2)*waveData/1024 + m_iBmpWidth/2;
		//iPosY = (y - m_iBmpHeight/2)*waveData/1024 + m_iBmpHeight/2;

		if (0 <= iPosX && iPosX < m_iBmpWidth &&
			0 <= iPosY && iPosY < m_iBmpHeight)
		{
			//�ֱ����ԭʼλͼ��iPosX��iPosY������Ⱦλͼ��x��y)��Ӧ����ʼλͼ����
			iPtrSource = iPosY * m_iBytesPerWidth + iPosX * 3;
			iPtrRender = y * m_iBytesPerWidth + x * 3;
			//��Ⱦλͼ�����´������
			for (int c = 0; c < 3; c++)
			{
				m_pBmpRender[iPtrRender + c] = m_pBmpSource[iPtrSource + c];
			}
		}

		lineIndex++;
	}
}

void ToCUDAWaveSpreadThreadStart(int* templpWave1, int* templpWave2, int m_iBmpWidth, int m_iBmpHeight) {


	CUDAWaveSpread << <16, 64 >> > (templpWave1, templpWave2, m_iBmpWidth, m_iBmpHeight);
	

	hipDeviceSynchronize();

	/*
	int* templpWave1;
	int* templpWave2;
	hipMallocManaged(&templpWave1, sizeof(int) * m_iBmpWidth * m_iBmpHeight);
	hipMallocManaged(&templpWave2, sizeof(int) * m_iBmpWidth * m_iBmpHeight);

	hipMemcpy(templpWave1, lpWave1, sizeof(int) * m_iBmpWidth * m_iBmpHeight, hipMemcpyHostToDevice);
	hipMemcpy(templpWave2, lpWave2, sizeof(int) * m_iBmpWidth * m_iBmpHeight, hipMemcpyHostToDevice);
	*/

	//testWaveSpread(lpWave1, lpWave2, m_iBmpWidth, m_iBmpHeight);

	//testWaveSpread << <1, 1 >> > (templpWave1, templpWave2, m_iBmpWidth, m_iBmpHeight);

	//hipDeviceSynchronize();
	/*
	hipMemcpy(lpWave2, templpWave1, sizeof(int) * m_iBmpWidth * m_iBmpHeight, hipMemcpyDeviceToHost);
	hipMemcpy(lpWave1, templpWave2, sizeof(int) * m_iBmpWidth * m_iBmpHeight, hipMemcpyDeviceToHost);
	*/

}
void ToCUDACUDAWaveRenderThreadStart(int* m_pWaveBuf1, BYTE* m_pBmpRender, BYTE* m_pBmpSource, int m_iBytesPerWidth, int m_iBmpWidth, int m_iBmpHeight) {
	CUDAWaveRender << <32, 64 >> > (m_pWaveBuf1,m_pBmpRender,m_pBmpSource,m_iBytesPerWidth,m_iBmpWidth,m_iBmpHeight);
	hipDeviceSynchronize();
}



