#include "hip/hip_runtime.h"
#include "CUDA.cuh"
#include <stdio.h>
#include <Windows.h>
__global__ void CUDAWaveSpread(int* AlpWave1, int*AlpWave2, int m_iBmpWidth, int m_iBmpHeight) {
	int Bid = blockIdx.x;
	int Tid = threadIdx.x;
	for (int i = m_iBmpWidth+ Bid*64*m_iBmpWidth+Tid * m_iBmpWidth; i < m_iBmpWidth + Bid * 64*m_iBmpWidth + Tid * m_iBmpWidth + m_iBmpWidth; i++)
	{
		//������ɢ
		AlpWave2[i] = ((AlpWave1[i - 1] + AlpWave1[i - m_iBmpWidth] +
			AlpWave1[i + 1] + AlpWave1[i + m_iBmpWidth]) >> 1) - AlpWave2[i];

		//����˥��
		AlpWave2[i] -= (AlpWave2[i] >> 5);
	}
	
	
}

__global__ void CUDAWaveRender(int* AlpWave1,  BYTE* tempM_pBmpRender, BYTE* tempM_pBmpSource, int m_iBytesPerWidth, int m_iBmpWidth, int m_iBmpHeight) {
	int iPtrSource = 0;
	int iPtrRender = 0;
	int iPosX = 0;
	int iPosY = 0;
	int y= blockIdx.x*64+threadIdx.x+1;
	int lineIndex = y * m_iBmpWidth;
	for (int x = 0; x < m_iBmpWidth; x++)
	{
		//���ݲ�������λͼ����ƫ��ֵ����Ⱦ�㣨x��y)��ӦԭʼͼƬ��iPosX��iPosY��
		iPosX = x + (AlpWave1[lineIndex - 1] - AlpWave1[lineIndex + 1]);
		iPosY = y + (AlpWave1[lineIndex - m_iBmpWidth] - AlpWave1[lineIndex + m_iBmpWidth]);
		//����һ�ּ���ƫ�Ƶķ���
		//int waveData = (1024 - m_pWaveBuf1[lineIndex]);
		//iPosX = (x - m_iBmpWidth/2)*waveData/1024 + m_iBmpWidth/2;
		//iPosY = (y - m_iBmpHeight/2)*waveData/1024 + m_iBmpHeight/2;

		if (0 <= iPosX && iPosX < m_iBmpWidth &&
			0 <= iPosY && iPosY < m_iBmpHeight)
		{
			//�ֱ����ԭʼλͼ��iPosX��iPosY������Ⱦλͼ��x��y)��Ӧ����ʼλͼ����
			iPtrSource = iPosY * m_iBytesPerWidth + iPosX * 3;
			iPtrRender = y * m_iBytesPerWidth + x * 3;
			//��Ⱦλͼ�����´������
			for (int c = 0; c < 3; c++)
			{
				tempM_pBmpRender[iPtrRender + c] = tempM_pBmpSource[iPtrSource + c];
				///Ͷ�벨Դ��ͼƬ��Ⱦ�Լ����㲨�����̲߳��У����������޸�tempM_pBmpSource��ֵ������ԭ��δ֪������tempM_pBmpSource�ظ���ֵ�ݱ�
			}
		}

		lineIndex++;
	}
}

__global__ void ModifyCUDALpWave(int* AlpWave1, int m_iBmpWidth, int m_iBmpHeight,int x, int y, int stoneSize, int stoneWeight) {
	int posX = 0;
	int posY = 0;
	for (int i = -stoneSize; i < stoneSize; i++)
	{
		for (int j = -stoneSize; j < stoneSize; j++)
		{
			posX = x + i;
			posY = y + j;

			//���Ʒ�Χ�����ܳ���ͼƬ
			if (posX < 0 || posX >= m_iBmpWidth ||
				posY < 0 || posY >= m_iBmpHeight)
			{
				continue;
			}
			//��һ��Բ�������ڣ���ʼ�����ܻ�����1
			if (i * i + j * j <= stoneSize * stoneSize)
			{
				AlpWave1[posY * m_iBmpWidth + posX] = stoneWeight;
			}

		}
	}
}

void ToCUDAWaveSpreadThreadStart(int* AlpWave1, int* AlpWave2, int m_iBmpWidth, int m_iBmpHeight) {
	CUDAWaveSpread << <16, 64 >> > (AlpWave1, AlpWave2, m_iBmpWidth, m_iBmpHeight);
	hipDeviceSynchronize();
}
void ToCUDACUDAWaveRenderThreadStart(int* AlpWave1, BYTE* tempM_pBmpRender, BYTE* tempM_pBmpSource, int m_iBytesPerWidth, int m_iBmpWidth, int m_iBmpHeight) {
	CUDAWaveRender << <32, 64 >> > (AlpWave1, tempM_pBmpRender, tempM_pBmpSource,m_iBytesPerWidth,m_iBmpWidth,m_iBmpHeight);
	hipDeviceSynchronize();
}
void ToModifyCUDALpWaveThreadStart(int* AlpWave1, int m_iBmpWidth, int m_iBmpHeight, int x, int y, int stoneSize, int stoneWeight) {
	ModifyCUDALpWave << <1, 1 >> > (AlpWave1, m_iBmpWidth, m_iBmpHeight, x, y, stoneSize, stoneWeight);
}


